#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#define N 32 * 1024 * 1024
#define imin(a,b) (a<b?a:b)
#define THREADS_PER_BLOCK 256

const int BLOCKS_PER_GRID = imin(32, (N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK);

__global__ void dotMul(float *a, float *b, float *c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    __shared__ float caches[THREADS_PER_BLOCK];
    
    float temp = 0;
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    caches[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            caches[cacheIndex] += caches[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0) {
        c[blockIdx.x] = caches[0];
    }
}

float malloc_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, *c;
    float result;
    float *deva, *devb, *devc;
    float elapsedT;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (float*)malloc(size * sizeof(float));
    b = (float*)malloc(size * sizeof(float));
    c = (float*)malloc(size * sizeof(float));

    hipMalloc((void**)&deva, size * sizeof(float));
    hipMalloc((void**)&devb, size * sizeof(float));
    hipMalloc((void**)&devc, size * sizeof(float));

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i + 2;
    }

    hipEventRecord(start, 0);

    hipMemcpy(deva, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devb, b, size * sizeof(float), hipMemcpyHostToDevice);

    dotMul<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(deva, devb, devc, size);

    hipMemcpy(c, devc, size * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedT, start, stop);
    
    result = 0.0f;
    for (int i = 0; i < size; i++) {
        result += c[i];
    }

    hipFree(deva);
    hipFree(devb);
    hipFree(devc);
    free(a);
    free(b);
    free(c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Value calculated: %3.6f\n", result);
    return elapsedT;
}

float cuda_host_alloc_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, *c;
    float result;
    float *deva, *devb, *devc;
    float elapsedT;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**)&c, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i + 2;
    }

    hipHostGetDevicePointer(&deva, a, 0);
    hipHostGetDevicePointer(&devb, b, 0);
    hipHostGetDevicePointer(&devc, c, 0);

    hipEventRecord(start, 0);

    dotMul<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(deva, devb, devc, size);

    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedT, start, stop);
    
    result = 0.0f;
    for (int i = 0; i < size; i++) {
        result += c[i];
    }

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Value calculated: %3.6f\n", result);
    return elapsedT;
}

float cuda_host_alloc_H2D_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, *c;
    float result;
    float *deva, *devb, *devc;
    float elapsedT;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
    c = (float*)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i + 2;
    }

    hipHostGetDevicePointer(&deva, a, 0);
    hipHostGetDevicePointer(&devb, b, 0);
    hipMalloc((void**)&devc, size * sizeof(float));

    hipEventRecord(start, 0);

    dotMul<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(deva, devb, devc, size);

    hipDeviceSynchronize();

    hipMemcpy(c, devc, size * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedT, start, stop);
    
    result = 0.0f;
    for (int i = 0; i < size; i++) {
        result += c[i];
    }

    hipHostFree(a);
    hipHostFree(b);
    free(c);
    hipFree(devc);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Value calculated: %3.6f\n", result);
    return elapsedT;
}

int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (prop.canMapHostMemory != 1) {
        printf("Device can't map memory!\n");
        return 0;
    }
    printf("Cuda Integreted: %s\n", (prop.integrated == 1)?"True":"False");

    hipSetDeviceFlags(hipDeviceMapHost);
    
    float elapsedTNoMapped = malloc_test(N);
    printf("Elapsed No Mapped: %3.6f ms\n", elapsedTNoMapped);
    
    float elapsedTMapped = cuda_host_alloc_test(N);
    printf("Elapsed Mapped: %3.6f ms\n", elapsedTMapped);

    float elapsedTMappedH2D = cuda_host_alloc_H2D_test(N);
    printf("Elapsed Mapped H2D: %3.6f ms\n", elapsedTMappedH2D);

    return 0;
}
