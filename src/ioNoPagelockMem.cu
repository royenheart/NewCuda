#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define SIZE (1024 * 1024) 

void malloc_test(bool up = true) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *host, *dev;

    host = (float*)malloc(sizeof(float) * SIZE);
    hipMalloc((void**)&dev, sizeof(float) * SIZE);

    int N = 100;
    if (up) {
        for (int i = 0; i < N; i++) {
            hipMemcpy(dev, host, sizeof(float) * SIZE, hipMemcpyHostToDevice);
        }
    } else {
        for (int i = 0; i < N; i++) {
            hipMemcpy(host, dev, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedT;
    hipEventElapsedTime(&elapsedT, start, stop);

    printf("Elapsed Time total: %.6f ms\n", elapsedT);
    printf("%s Transport speed: %.6fMB/s\n", (up)?"H2D":"D2H",(float)(100 * sizeof(float) * SIZE) / 1024.0f / 1024.0f * 1000.0f / elapsedT);

    free(host);
    hipFree(dev);
    hipFree(start);
    hipFree(stop);
}

int main() {
    malloc_test(true);
    malloc_test(false);

    return 0;
}