#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#ifdef _WIN32
    #include <time.h>
    #include <sys/timeb.h>
#else
    #include <sys/time.h>
    #include <sys/timeb.h>
#endif

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)
#define INT_N (N * sizeof(int))
#define INT_FULL_DATA_SIZE (FULL_DATA_SIZE * sizeof(int))

using namespace std;

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2.0f;
    }
}

int main(int argc, char* argv[]) {
    struct timeb timeSeed;
    ftime(&timeSeed);
    srand(timeSeed.time * 1000 + timeSeed.millitm);

    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        cout << "Device can't use device overlap, exit!" << endl;
        return 0;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int *hosta, *hostb, *hostc;
    int *deva1, *deva2, *devb1, *devb2, *devc1, *devc2;

    hipMalloc((void**)&deva1, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devb1, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&deva2, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devb2, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devc1, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devc2, INT_FULL_DATA_SIZE);

    hipHostAlloc((void**)&hosta, INT_FULL_DATA_SIZE, hipHostMallocDefault);
    hipHostAlloc((void**)&hostb, INT_FULL_DATA_SIZE, hipHostMallocDefault);
    hipHostAlloc((void**)&hostc, INT_FULL_DATA_SIZE, hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        hosta[i] = rand();
        hostb[i] = rand();
    }

    for (int i = 0; i < FULL_DATA_SIZE; i += 2 * N) {
        hipMemcpyAsync(deva1, hosta + i, INT_N, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(deva2, hosta + i + N, INT_N, hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(devb1, hostb + i, INT_N, hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(devb2, hostb + i + N, INT_N, hipMemcpyHostToDevice, stream2);
        kernel<<<N / 256, 256, 0, stream1>>>(deva1, devb1, devc1);
        kernel<<<N / 256, 256, 0, stream2>>>(deva2, devb2, devc2);
        hipMemcpyAsync(hostc + i, devc1, INT_N, hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(hostc + i + N, devc2, INT_N, hipMemcpyDeviceToHost, stream2);
    }

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedT; 
    hipEventElapsedTime(&elapsedT, start, stop);
    printf("Time taken: %3.6f ms\n", elapsedT);

    hipHostFree(hosta);
    hipHostFree(hostb);
    hipHostFree(hostc);
    hipFree(deva1);
    hipFree(devb1);
    hipFree(devc1);
    hipFree(deva2);
    hipFree(devb2);
    hipFree(devc2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}