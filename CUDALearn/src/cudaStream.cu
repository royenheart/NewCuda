#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#ifdef _WIN32
    #include <time.h>
    #include <sys/timeb.h>
#else
    #include <sys/time.h>
    #include <sys/timeb.h>
#endif

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)
#define INT_N (N * sizeof(int))
#define INT_FULL_DATA_SIZE (FULL_DATA_SIZE * sizeof(int))

using namespace std;

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2.0f;
    }
}

int main(int argc, char* argv[]) {
    struct timeb timeSeed;
    ftime(&timeSeed);
    srand(timeSeed.time * 1000 + timeSeed.millitm);

    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        cout << "Device can't use device overlap, exit!" << endl;
        return 0;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *hosta, *hostb, *hostc;
    int *deva, *devb, *devc;

    hipMalloc((void**)&deva, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devb, INT_FULL_DATA_SIZE);
    hipMalloc((void**)&devc, INT_FULL_DATA_SIZE);

    hipHostAlloc((void**)&hosta, INT_FULL_DATA_SIZE, hipHostMallocDefault);
    hipHostAlloc((void**)&hostb, INT_FULL_DATA_SIZE, hipHostMallocDefault);
    hipHostAlloc((void**)&hostc, INT_FULL_DATA_SIZE, hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        hosta[i] = rand();
        hostb[i] = rand();
    }

    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        hipMemcpyAsync(deva, hosta + i, INT_N, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(devb, hostb + i, INT_N, hipMemcpyHostToDevice, stream);
        kernel<<<N / 256, 256, 0, stream>>>(deva, devb, devc);
        hipMemcpyAsync(hostc + i, devc, INT_N, hipMemcpyDeviceToHost, stream);
    }

    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedT; 
    hipEventElapsedTime(&elapsedT, start, stop);
    printf("Time taken: %3.6f ms\n", elapsedT);

    hipHostFree(hosta);
    hipHostFree(hostb);
    hipHostFree(hostc);
    hipFree(deva);
    hipFree(devb);
    hipFree(devc);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);

    return 0;
}